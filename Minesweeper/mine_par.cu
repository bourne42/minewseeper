#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <map>
#include <vector>
#include <iostream>
#include <cmath>

#include <stdlib.h>     /* srand, rand */
#include <time.h> 

#include "CycleTimer.h"

#define IDX(x,y) ((x) + (width * (y)))
#define GRID(x,y) grid[((x) + (width * (y)))]
#define BOMB(x,y) (GRID((x),(y)) == 10 || GRID((x),(y)) == (-10))
#define HIDDEN(x,y) (GRID((x),(y)) < 0)
#define IX(i) ((i)%width)
#define IY(i) ((i)/width)


void printGrid(int width, int height, int *grid) {
  int i,j;
  std::cout<<"   ";
  for(i=0; i<width; i++)
    std::cout<<(i%10)<<"   ";
  std::cout<<"\n";

  for(j=0; j<height; j++) {
    std::cout<<(j%10)<<" ";
    for(i=0; i<width; i++) {
      if(grid[IDX(i,j)]==0){
	std::cout<<"    ";
        continue;
      }
      std::cout<<grid[IDX(i,j)]<<" ";
      if(grid[IDX(i,j)]>=0)
	std::cout<<" ";
      if(abs(grid[IDX(i,j)])<10)
	std::cout<<" ";
    }
    std::cout<<"\n";
  }
  std::cout<<"   ";
  for(i=0; i<width; i++)
    std::cout<<(i%10)<<"   ";
  std::cout<<"\n";
}

void printGridCompact(int width, int height, int *grid) {
  int i,j;
  std::cout<<"  ";
  for(i=0; i<width; i++)
    std::cout<<(i%10)<<" ";
  std::cout<<"\n";

  for(j=0; j<height; j++) {
    std::cout<<(j%10)<<" ";
    for(i=0; i<width; i++) {
      if(grid[IDX(i,j)]==0){
	std::cout<<"  ";
        continue;
      }
      std::cout<<grid[IDX(i,j)]<<" ";
    }
    std::cout<<"\n";
  }
  std::cout<<"  ";
  for(i=0; i<width; i++)
    std::cout<<(i%10)<<"   ";
  std::cout<<"\n";
}


/*
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {
  
  // compute overall index from position of thread in current block,
  // and given the block we are in
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (index < N)
    result[index] = alpha * x[index] + y[index];
}
*/

/**
 * If on a border will set to index number, 0 otherwise. 
 * If the cell is on the grid border but not on a real border will be negative
 */
__global__ void identify_boundaries(int width, int height, int *grid, int *borders) {
  //int index = blockIdx.x * blockDim.x + threadIdx.x;
  //int index = threadIdx.x;
  //int x = IX(index), y = IY(index);
  int x = threadIdx.x, y = blockIdx.x;
  int index = x + (y*width);
  int i,j;

  borders[index] = 0;

  if(!HIDDEN(x,y)) 
    return;

  if(x==0 || x==(width-1) || y==0 || y==(height-1)) {
    borders[index] = -(index + 1);
    //return;
  }

  for(i=max(0, x-1); i<=min(width-1, x+1); i++) {
    for(j=max(0, y-1); j<=min(height-1, y+1) ; j++) {
      if(!HIDDEN(i,j)) {
	borders[index] = index + 1;
	return;
      }
    }
  }

}

/**
 * If a tile is on a boundary it will attempty to make itself a smaller boundary
 *   by looking at it's neighbors. If it finds a smaller neighbor then set done to true.
 * If the tile is on the grid border but not the solving border then the value will be negative.
 */
__global__ void consolidate_boundaries(int width, int height, 
				       int *grid, int *borders, bool *done) {
  //int index = threadIdx.x;
  //int x = IX(index), y = IY(index);
  int x = threadIdx.x, y = blockIdx.x;
  int index = x + (y*width);

  int i,j;
  int newMin = borders[index];
  int other;

  bool border = newMin < 0;
  if(border)
    newMin = -newMin;

  for(j=max(0, y-2); j<=min(height-1, y+2); j++) {
    for(i=max(0, x-2); i<=min(width-1, x+2); i++) {
      other = abs(borders[IDX(i,j)]);
      if(other != 0 && other<newMin)
	newMin = other;
    }
  }

  if(border)
    newMin = -newMin;

  if(newMin != borders[index]) {
    borders[index] = newMin;
    *done = false;
  }
}

/**
 * Every revealed cell next to a border cell will become negative of that border id.
 * All other cells will become 0 (except other border cells). 
 * When called *border should have positive values on borders, negative on grid
 *   borders, and 0 otherwise.
 */
__global__ void update_borders(int width, int height, int *border, int *grid, bool *flags) {
  //int index = threadIdx.x;
  //int x = IX(index), y = IY(index);
  int x = threadIdx.x, y = blockIdx.x;
  int index = x + (y*width);

  int i, j, xt, yt;

  // if a cell is flagged then automatically becomes 0
  if(flags[index]) {
    border[index] = 0;
    return;
  }

  // if on a border do nothing
  if(border[index] > 0) {
    return;
  }

  // if a cell is negative then it was on a grid border and not an actual border
  // if hidden then don't do anything with the cell
  if(border[index]<0 || HIDDEN(x,y)) {
    border[index] = 0;
    return;
  }

  // each known cell tries to identify with a border
  for(j=-1; j<=1; j++) {
    yt = y+j;
    if(yt<0 || yt>=height)
      continue;
    for(i=-1; i<=1; i++) {
      xt = x+i;
      if(xt<0 || xt>=width)
	continue;
      if(border[IDX(xt,yt)]>0 && !flags[IDX(xt,yt)]) {
	border[index] = -border[IDX(xt,yt)];
	return;
      }
    }
  }  
}

/**
 * sets all border cells to 0
 */
__global__ void grid_borders_zero(int width, int height, int *arr) {
  //int index = threadIdx.x;
  int index = threadIdx.x + (width*blockIdx.x);
  arr[index]=0;
}

/**
 * Sets all cells that are 3 to 0
 */
__global__ void finalize_knowns(int width, int height, int *knowns) {
  //int index = threadIdx.x;
  int index = threadIdx.x + (width*blockIdx.x);
  if(knowns[index]==3)
    knowns[index]=0;
}


/**
 * Sets all cells that are uncovered or flagged to 0
 */
__global__ void clean_knowns(int width, int height, int *grid, bool *flags, int *knowns) {
  //int index = threadIdx.x;
  int index = threadIdx.x + (width*blockIdx.x);
  if(flags[index] || grid[index]>=0)
    knowns[index]=0;
}

/**
 * Fill in the temp grid as follows: positive number of unknown bombs if uncovered. 
 * Subtract known (flagged) bombs from self (if positive number). 
 * 0 otherwise
 */
__global__ void make_temp_grid(int width, int height, int *grid, bool *flags, 
			       int *old_knowns, int *temp_grid) {
  //int index = threadIdx.x;
  //int x = IX(index), y = IY(index);
  int x = threadIdx.x, y = blockIdx.x;
  int index = x + (y*width);

  int i, j, xt, yt;

  if(flags[index] || grid[index]<0) {
    temp_grid[index] = 0;
    return;
  }

  int grid_num = grid[index];

  for(j=-1; j<=1; j++) {
    yt = y+j;
    if(yt<0 || yt>=height)
      continue;
    for(i=-1; i<=1; i++) {
      xt = x+i;
      if(xt<0 || xt>=width)
	continue;
      if(flags[IDX(xt,yt)] || old_knowns[IDX(xt,yt)]==1)
	grid_num--;
    }
  }

  temp_grid[index] = grid_num;
}

/**
 * Uses values in blockId and threadId to know where bombs are placed
 *   in this call. The combined 4 ints can be seen as one long bit vector
 *   which store where bombs are, index will return true if there is a 1 (bomb)
 *   at that index
 */
__device__ bool is_bomb(int index, int thread_max, int x_block_max, int y_block_max) {
  // only need threadIdx, if index<10 look at the first 1024 values which are stored
  //   in threadIdx.x
  if(index < thread_max) {
    return ((threadIdx.x>>index) & 1) == 1;
  }

  // there's probably a cleaner method to do this, but its pretty short and not necessarily slow
  index -= thread_max;
  if(index < x_block_max) {
    return ((blockIdx.x>>index) & 1) == 1;
  }

  index -= x_block_max;
  if(index < y_block_max) {
    return ((blockIdx.y>>index) & 1) == 1;
  }

  index -= y_block_max;
  //if(index < max_block_dim) {
    return ((blockIdx.z>>index) & 1) == 1;
    //}
    //return true;
}

/**
 * Each thread takes one possible solution, as defined by threadIdx and blockIdx. 
 * If the solution is not valid do nothing. 
 * All values in knowns should be 0 at start of the call. 
 * If the value can be a bomb the first bit will be 1, else 0. 
 * If the value can be empty the second bit will be 1, else 0. 
 */
__global__ void find_solvable_tiles(int width, int height, int *grid, int *knowns, 
				    int *border, int border_size, int *temp_grid, 
				    int *known_border, int known_border_size, 
				    int thread_max, int x_block_max, int y_block_max) {
  // in the for loop keeps track of which known cell we are checking against
  int c, i, j, x, y, cell, bombs;

  for(c = 0; c<known_border_size; c++) { 
    cell = known_border[c];
    bombs = 0;
    x = IX(cell);
    y = IY(cell);

    for(i=max(x-1,0); i<=min(x+1,width-1); i++) {
      for(j=max(y-1,0); j<=min(y+1,height-1); j++) {
	if(temp_grid[IDX(i,j)]<0)
	  if(is_bomb((-1-temp_grid[IDX(i,j)]), thread_max, x_block_max, y_block_max))
	    bombs++;
      }
    }

    if(temp_grid[cell]!=bombs)
      return;
  }

  // if the program gets here then this is a valid solution
  // commit into knowns (first bit->1 if bomb, second bit->1 if not)
  for(c=0; c<border_size; c++) {
    cell = border[c];
    //atomicAdd(&knowns[cell], 1);
    if(is_bomb(c, thread_max, x_block_max, y_block_max)) {
      atomicOr(&knowns[cell], 1);
    } else {
      atomicOr(&knowns[cell], 2);
    }
  }
}

/**
 * Scans ever element in the border if it has solution
 * Callable right after find_solvable_tiles with same data
 */
__global__ void border_has_solution(int width, int *knowns, int *border, 
				    int border_id, bool *solution) {
  //int index = threadIdx.x;
  int index = threadIdx.x + (width * blockIdx.x);
  if(border[index] != border_id)
    return;
  int c = knowns[index];
  if(c==1 || c==2)
    *solution = true;
}

__global__ void preliminary_knowns(int width, int height, int *grid, bool *flags, 
				   int *border, int *knowns) {
  //int index = threadIdx.x;
  int x = threadIdx.x, y = blockIdx.x;
  int index = x + (y*width);

  if(index >= (width*height) || border[index]>=0)
    return;
  //int x = IX(index), y = IY(index);
  int bomb_count = 0, unknown_count = 0;

  for(int j=max(0,y-1); j<=min(height-1, y+1); j++) 
    for(int i=max(0,x-1); i<=min(width-1, x+1); i++) {
      int c = IDX(i,j);
      if(knowns[c]==1 || flags[c])
	bomb_count++;
      else if(knowns[c]==0 && grid[c]<0) 
	unknown_count++;
    }

  if(unknown_count == 0)
    return;

  int all_bombs = 0; // will be 1 if all should be bombs, -1 if clear, 0 ow
  if((grid[index]-bomb_count) == unknown_count)
    all_bombs = 1;
  else if(bomb_count == grid[index])
    all_bombs = -1;
  
  // if know neighbors then fill them
  if(all_bombs!=0) {
    for(int j=max(0,y-1); j<=min(height-1, y+1); j++) 
      for(int i=max(0,x-1); i<=min(width-1, x+1); i++) {
	int c = IDX(i,j);
	if(flags[c] || grid[c]>=0 || border[c]<=0 || knowns[c]!=0)
	  continue;
	if(all_bombs<0)//all clear
	  atomicOr(&knowns[c], 2);
	else //all bombs
	  atomicOr(&knowns[c], 1);
      }
  }
}

/**
 * Recursive call to display cell
 */
void hint_display_cell(int x, int y, int width, int height, int *grid) {
  if(GRID(x,y)>=0)
    return;

  grid[IDX(x,y)] = -(GRID(x,y)+1);

  if(GRID(x,y)==0) {
    for(int i = std::max(x-1,0); i<=x+1 && i<width; i++) {
      for(int j = std::max(y-1,0); j<=y+1 && j<height; j++) {
        if(HIDDEN(i,j))
          hint_display_cell(i,j, width, height, grid);
      }
    }
  }
}

/**
 * Knowns is the working array that this function will overwrite with new data. 
 * Assumes that all flags are correct. 
 */
bool known_cells(int width, int height, int *grid, bool *flags, int *knowns, 
		 int* border, bool random_sol, bool display_time, bool display_knowns,
		 bool display_debug_grids) {
  double totalStartTime = CycleTimer::currentSeconds();

  hipDeviceProp_t *prop = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t));
  hipGetDeviceProperties(prop, 0);
  //std::cout<<"Threads: "<<prop->maxThreadsPerBlock<<" Blocks: "<<prop->maxGridSize[0]<<
  //" "<<prop->maxGridSize[1]<<" "<<prop->maxGridSize[2]<<"\n";

  bool changed_grid = false;

  int size = width*height;

  //int *border = new int[size];
  int *device_border;

  int *device_grid;
  bool *device_flags;
  int *device_knowns;
  int *device_temp_grid;

  bool *device_border_done;
  bool *device_solution;

  int bool_array_size = sizeof(bool)*size;
  int int_array_size = sizeof(int)*size;

  hipMalloc(&device_grid, int_array_size);
  hipMalloc(&device_flags, bool_array_size);
  hipMalloc(&device_knowns, int_array_size);
  hipMalloc(&device_temp_grid, int_array_size);
  hipMalloc(&device_border, int_array_size);

  hipMalloc(&device_border_done, sizeof(bool));
  hipMalloc(&device_solution, sizeof(bool));

  hipMemcpy(device_grid, grid, int_array_size, hipMemcpyHostToDevice);
  hipMemcpy(device_knowns, knowns, int_array_size, hipMemcpyHostToDevice);
  hipMemcpy(device_flags, flags, bool_array_size, hipMemcpyHostToDevice);

  // each cell identifies itself as border or grid boundary
  identify_boundaries<<<height, width>>>(width, height, device_grid, device_border);
  hipDeviceSynchronize();
  
  bool border_done = false;

  // make all numbers on the same border the same (lowest) number
  while(!border_done) {
    border_done = true;
    hipMemcpy(device_border_done, &border_done, sizeof(bool), hipMemcpyHostToDevice);
    consolidate_boundaries<<<height, width>>>(width, height, device_grid, 
					device_border, device_border_done);
    hipDeviceSynchronize();
    hipMemcpy(&border_done, device_border_done, 
	       sizeof(bool), hipMemcpyDeviceToHost);
  }

  //hipDeviceSynchronize();
  update_borders<<<height, width>>>(width, height, device_border, device_grid, device_flags);
  hipDeviceSynchronize();
  // borders are now identified by numbers
  hipMemcpy(border, device_border, int_array_size, hipMemcpyDeviceToHost);

  if(display_debug_grids) {
    std::cout<<"\nBorder:\n";
    printGrid(width, height, border);
  }

  // start filling out temp grid
  // uncovered cells in device_temp_grid will now have the number of unflagged mines touching them
  clean_knowns<<<height, width>>>(width, height, device_grid, device_flags, device_knowns);
  hipDeviceSynchronize();


  double prelimStartTime = CycleTimer::currentSeconds();
  preliminary_knowns<<<height, width>>>(width, height, device_grid, device_flags, 
				  device_border, device_knowns);
  hipDeviceSynchronize();
  preliminary_knowns<<<height, width>>>(width, height, device_grid, device_flags, 
					device_border, device_knowns);
  hipDeviceSynchronize();
  double prelimEndTime = CycleTimer::currentSeconds();

  if(display_debug_grids) {
    hipMemcpy(knowns, device_knowns, int_array_size, hipMemcpyDeviceToHost);
    std::cout<<"After prelim:\n";
    printGrid(width, height, knowns);
  }

  make_temp_grid<<<height, width>>>(width, height, device_grid, device_flags, 
			      device_knowns, device_temp_grid);
  hipDeviceSynchronize();
  
  hipMemcpy(knowns, device_knowns, int_array_size, hipMemcpyDeviceToHost);

  int *temp_grid = new int[size];
  // get the temp grid from CPU
  hipMemcpy(temp_grid, device_temp_grid, int_array_size, hipMemcpyDeviceToHost);

  std::map<int, std::vector<int> > borders;
  std::map<int, std::vector<int> > known_borders;

  // fill in the lists for the borders
  // One list for the known and another for the unknown borders
  for(int i=0; i<size; i++) {
    // if border is 0 or there is a previous solution, don't be part of the border
    if(border[i]==0 || knowns[i]!=0)
      continue;
    // fill in vector for the known boundary
    if(border[i]<0) {
      if(known_borders.count(-border[i])==0)
	known_borders[-border[i]] = std::vector<int>(1,i);
      else
	known_borders[-border[i]].push_back(i);
      continue;
    }

    if(borders.count(border[i])==0)
      borders[border[i]] = std::vector<int>(1,i);
    else
      borders[border[i]].push_back(i);
    // temp grid will have the -index-1 when the cell has an unknown, will index into border array
    temp_grid[i] = -(borders[border[i]].size());
  }

  // changed temp_grid above, now put back into device_temp and delete
  hipMemcpy(device_temp_grid, temp_grid, int_array_size, hipMemcpyHostToDevice);
  if(display_debug_grids) {
    std::cout<<"\nTemp Grid:\n";
    printGrid(width, height, temp_grid);
  }
  delete(temp_grid);

  // Calculate the max number of cells each dimension can be responsible for.
  // Do this by calculating log_2 of the max thread and difference max dimensions
  //   based on GPU information
  int thread_max = 0, x_block_max = 0, y_block_max = 0, z_block_max = 0;
  int temp = prop->maxThreadsPerBlock;
  while (temp >>= 1) ++thread_max;
  temp = prop->maxGridSize[0];
  while (temp >>= 1) ++x_block_max;
  temp = prop->maxGridSize[1];
  while (temp >>= 1) ++y_block_max;
  temp = prop->maxGridSize[2];
  while (temp >>= 1) ++z_block_max;
  z_block_max--;
  x_block_max--;
  y_block_max--;
  if(display_time) 
    std::cout<<"Max Threads: "<<thread_max<<" Max Blocks: "<<x_block_max<<" "<<
      y_block_max<<" "<<z_block_max<<"\n";
  //std::cout<<prop->maxThreadsPerBlock<<" "<<prop->maxGridSize[0]<<" "<<
  //prop->maxGridSize[1]<<" "<<prop->maxGridSize[2]<<"\n";

  double startTime = CycleTimer::currentSeconds();
  double calcTime = 0;

  // for each boundary find possible solutions, actuall work here
  for (std::map<int,std::vector<int> >::iterator it=borders.begin(); 
       it!=borders.end(); ++it) {
    std::vector<int> border_vec = it->second;
    std::vector<int> known_border_vec = known_borders[it->first];

    /*
    std::cout<<"Border "<< it->first <<" size: "<<border_vec.size()<<
      " known size: "<<known_border_vec.size()<<"\nBorder: ";
    for(int i=0; i<border_vec.size(); i++)
      std::cout<<border_vec[i]<<" ";
    std::cout<<"\nKnown: ";
    for(int i=0; i<known_border_vec.size(); i++)
      std::cout<<known_border_vec[i]<<" ";
    std::cout<<"\n";
    */

    //std::cout << it->first << " => " << it->second << '\n';
    int *border_elements;
    hipMalloc(&border_elements, sizeof(int)*border_vec.size());
    hipMemcpy(border_elements, &border_vec[0], sizeof(int)*border_vec.size(), 
	       hipMemcpyHostToDevice);

    int *known_border_elements;
    hipMalloc(&known_border_elements, sizeof(int)*known_border_vec.size());
    hipMemcpy(known_border_elements, &known_border_vec[0], 
	       sizeof(int)*known_border_vec.size(), hipMemcpyHostToDevice);

    // these are the number of grid cells the different dimensions will represent
    // actual numbers that will be sent to the gpu will be 2^x
    int block_threads = thread_max;
    int block_x = 0;
    int block_y = 0;
    int block_z = 0;

    int border_left = border_vec.size();
    bool skip = false;

    if(border_left <= thread_max) {
      block_threads = border_left;
    } else {
      border_left -= thread_max;
      
      // set block_x
      if(border_left <= x_block_max) {
	block_x = border_left;
      } else {
	block_x = x_block_max;
	border_left -= x_block_max;

	// if still more left set block_y
	if(border_left <= y_block_max) {
	  block_y = border_left;
	} else {
	  block_y = y_block_max;
	  border_left -= y_block_max;
	  
	  // if still more left set block_z
	  if(border_left <= z_block_max) {
	    block_z = border_left;
	  } else {
	    //if you get to this point the border is quite large, just
	    // give up
	    skip = true;
	  }
	}
      }
    }
    
    if(!skip) {
      dim3 grid_block(1<<block_x, 1<<block_y, 1<<block_z);
      
      if(display_time)
		std::cout<<"Border size: "<<border_vec.size()<<"\nThreads: "<<
	  pow(2.0,block_threads)<< " Grid: "<<grid_block.x<<" "<<
	  grid_block.y<<" "<<grid_block.z<<"\n";

      double calcStartTime = CycleTimer::currentSeconds();
      find_solvable_tiles<<< grid_block,
		1<<block_threads >>>(width, height, device_grid, device_knowns, 
				  border_elements, border_vec.size(), device_temp_grid,
				  known_border_elements, known_border_vec.size(), 
				  thread_max, x_block_max, y_block_max);
      hipDeviceSynchronize();
      double calcEndTime = CycleTimer::currentSeconds();
      if(display_time)
	std::cout<<"Calc Time: "<<(calcEndTime-calcStartTime)<<"\n\n";
      calcTime += (calcEndTime-calcStartTime);

      // check if there is a solution for this border, if not unveil a random tile
      //random_sol = false;
      if(random_sol) {
	bool solution=false;
	hipMemcpy(device_solution, &solution, sizeof(bool), hipMemcpyHostToDevice);
	border_has_solution<<<height, width>>>(width, device_knowns, device_border, it->first, 
					       device_solution);
	hipDeviceSynchronize();
	hipMemcpy(&solution, device_solution, sizeof(bool), hipMemcpyDeviceToHost);
	if(!solution) {
	  //std::cout<<"NO SOLUTION for border "<< it->first <<"\n";
	  changed_grid = true;
	}
	srand(time(NULL));
	int c = 0;//don't want to repeat forever
	while(!solution && c<border_vec.size()) {
	  int t = border_vec[rand()%border_vec.size()];
	  int x = IX(t), y = IY(t);
	  if(!BOMB(x,y)) {
	    std::cout<<"Revealed grid "<<x<<" "<<y<<"\n";
	    hint_display_cell(x, y, width, height, grid);
	    solution = true;
	  }
	  c++;
	}
      }
    }

    // clean up
    hipFree(border_elements);
    hipFree(known_border_elements);
  }

  if(display_time) {
    std::cout<<"\nTotal Calc Time: "<<calcTime<<"\n";
    double endTime = CycleTimer::currentSeconds();
    std::cout<<"Prelim run time: "<<(prelimEndTime-prelimStartTime)<<
      "\nMain work loop: "<<(endTime-startTime)<<"\n";
  }
  //hipDeviceSynchronize();

  //hipMemcpy(knowns, device_knowns, int_array_size, hipMemcpyDeviceToHost);
  //std::cout<<"\nBefore Clean:\n";
  //printGrid(width,height,knowns);

  finalize_knowns<<<height, width>>>(width, height, device_knowns);
  clean_knowns<<<height, width>>>(width, height, device_grid, device_flags, device_knowns);
  hipDeviceSynchronize();

  hipMemcpy(knowns, device_knowns, int_array_size, hipMemcpyDeviceToHost);
  
  hipFree(device_grid);
  hipFree(device_flags);
  hipFree(device_knowns);
  hipFree(device_temp_grid);
  hipFree(device_border_done);
  hipFree(device_border);
  hipFree(device_solution);

  free(prop);
  delete(border);

  if(display_time) {
    double totalEndTime = CycleTimer::currentSeconds();
    std::cout<<"Total execution time: "<<(totalEndTime-totalStartTime)<<"\n";
  }

  if(display_knowns || display_debug_grids) {
    std::cout<<"\nFinal knowns:\n";
    printGridCompact(width,height,knowns);
    //std::cout<<"Done\n";
  }

  return changed_grid;
}



void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {
  /*
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //
    size_t size = sizeof(float)*N;
    hipMalloc(&device_x, size);
    hipMalloc(&device_y, size);
    hipMalloc(&device_result, size);

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, size, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, size, hipMemcpyHostToDevice);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    // run kernel
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();

    // end timing after result has been copied back into host memory

    double endTime = CycleTimer::currentSeconds();

    //
    // TODO copy result from GPU using hipMemcpy
    //
    hipMemcpy(resultarray, device_result, size, hipMemcpyDeviceToHost);

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
  */
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}
